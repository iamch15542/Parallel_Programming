#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(float stepX, float stepY, float lowerX, float lowerY, int* img_result, int maxIterations, int pitch) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    
    float x = lowerX + (blockIdx.x * blockDim.x + threadIdx.x) * stepX;
    float y = lowerY + (blockIdx.y * blockDim.y + threadIdx.y) * stepY;
    
    float z_re = x, z_im = y;
    int i;
    for (i = 0; i < maxIterations; ++i) {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = x + new_re;
        z_im = y + new_im;
    }
    int* idx = (int*)((char*)img_result + (blockIdx.y * blockDim.y + threadIdx.y) * pitch) + (blockIdx.x * blockDim.x + threadIdx.x);
    *idx = i;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    // Declare the host memory
    int *h_result;
    hipHostAlloc((void **)&h_result, resX * resY * sizeof(int), hipHostMallocDefault);

    // Declare the cuda memory
    int *c_result;
    size_t pitch;
    hipMallocPitch((void **)&c_result, &pitch, sizeof(int) * resX, resY);

    dim3 blockSize(16, 16);
    dim3 numBlock(resX / 16, resY / 16);

    mandelKernel<<<numBlock, blockSize>>>(stepX, stepY, lowerX, lowerY, c_result, maxIterations, pitch);

    // 等待 GPU 所有 thread 完成
    hipDeviceSynchronize();

    // 將 Device 的資料傳回給 Host
    hipMemcpy2D(h_result, resX * sizeof(int), c_result, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);

    for(int i = 0; i < resX * resY; ++i) {
        *(img+i) = *(h_result+i);
    }

    // free memory
    hipHostFree(h_result);
    hipFree(c_result);
}
