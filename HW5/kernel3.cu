#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(float stepX, float stepY, float lowerX, float lowerY, int* img_result, int maxIterations, int pitch, int groups) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    for(int cnt = 0; cnt < groups; ++cnt) {
        float x = lowerX + ((blockIdx.x * blockDim.x + threadIdx.x) * groups + cnt) * stepX;
        float y = lowerY + (blockIdx.y * blockDim.y + threadIdx.y) * stepY;
        
        float z_re = x, z_im = y;
        int i;
        for (i = 0; i < maxIterations; ++i) {
            if (z_re * z_re + z_im * z_im > 4.f)
                break;

            float new_re = z_re * z_re - z_im * z_im;
            float new_im = 2.f * z_re * z_im;
            z_re = x + new_re;
            z_im = y + new_im;
        }
        int* idx = (int*)((char*)img_result + (blockIdx.y * blockDim.y + threadIdx.y) * pitch) + (blockIdx.x * blockDim.x + threadIdx.x) * groups + cnt;
        *idx = i;
    }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    // Declare the host memory
    int *h_result;
    hipHostAlloc((void **)&h_result, resX * resY * sizeof(int), hipHostMallocDefault);

    // Declare the cuda memory
    int *c_result, groups = 5;
    size_t pitch;
    hipMallocPitch((void **)&c_result, &pitch, sizeof(int) * resX, resY); // 4 * 1600 = 6400 -> pitch = 6656

    dim3 blockSize(16, 16);
    dim3 numBlock(resX / 80, resY / 16);

    mandelKernel<<<numBlock, blockSize>>>(stepX, stepY, lowerX, lowerY, c_result, maxIterations, pitch, groups);

    // 等待 GPU 所有 thread 完成
    hipDeviceSynchronize();

    // 將 Device 的資料傳回給 Host
    hipMemcpy2D(h_result, resX * sizeof(int), c_result, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);

    for(int i = 0; i < resX * resY; ++i) {
        *(img+i) = *(h_result+i);
    }

    // free memory
    hipHostFree(h_result);
    hipFree(c_result);
}
