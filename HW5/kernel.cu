#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(float stepX, float stepY, float lowerX, float lowerY, int* img_result, int maxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    __shared__ int block_result[256];
    
    float x = lowerX + (blockIdx.x * blockDim.x + threadIdx.x) * stepX;
    float y = lowerY + (blockIdx.y * blockDim.y + threadIdx.y) * stepY;
    
    float z_re = x, z_im = y;
    int i;
    for (i = 0; i < maxIterations; ++i) {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = x + new_re;
        z_im = y + new_im;
    }

    block_result[threadIdx.y * blockDim.x + threadIdx.x] = i;
    __syncthreads();

    if(threadIdx.x == 0 && threadIdx.y == 0) {
        for(int i = 0; i < 16; ++i) {
            for(int j = 0; j < 16; ++j) {
                int idx = (blockIdx.y * blockDim.y + j) * (gridDim.x * blockDim.x) + (blockIdx.x * blockDim.x + i);
                img_result[idx] = block_result[j * 16 + i];
            }
        }
    }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    // Declare the host memory
    // int *h_result = (int *)malloc(resX * resY * sizeof(int));

    // Declare the cuda memory
    int *c_result;
    hipMalloc(&c_result, resX * resY * sizeof(int));

    dim3 blockSize(16, 16);
    dim3 numBlock(resX / 16, resY / 16);

    mandelKernel<<<numBlock, blockSize>>>(stepX, stepY, lowerX, lowerY, c_result, maxIterations);

    // 將 Device 的資料傳回給 Host
    hipMemcpy(img, c_result, resX * resY * sizeof(int), hipMemcpyDeviceToHost);

    // free memory
    hipFree(c_result);
}
