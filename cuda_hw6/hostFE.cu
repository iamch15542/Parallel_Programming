#include "hip/hip_runtime.h"
#include "hostFE.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void convolution(float *inputImage, float *filter, float *outputImage, int filterWidth, int imageHeight, int imageWidth) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // col
    int j = blockIdx.y * blockDim.y + threadIdx.y; // row

    int sum = 0;
    int halfwidth = filterWidth / 2;
    // Apply the filter to the neighborhood
    for(int fi = -halfwidth; fi <= halfwidth; ++fi) {
        for(int fj = -halfwidth; fj <= halfwidth; ++fj) {
            if(i + fi >= 0 && i + fi < imageHeight && j + fj >= 0 && j + fj < imageWidth && filter[(fi + halfwidth) * filterWidth + fj + halfwidth]) {
                sum += inputImage[(i + fi) * imageWidth + j + fj] * filter[(fi + halfwidth) * filterWidth + fj + halfwidth];
            }
        }
    }
    outputImage[i * imageWidth + j] = sum;
}

void hostFE(int filterWidth, float *filter, int imageHeight, int imageWidth,
            float *inputImage, float *outputImage)
{
    // Declare the cuda memory
    int data_size = imageHeight * imageWidth * sizeof(float);

    float *d_inputImage;
    hipMalloc(&d_inputImage, data_size);

    float *d_filter;
    hipMalloc(&d_filter, filterWidth * filterWidth * sizeof(float));

    float *img_result;
    hipMalloc(&img_result, data_size);

    // copy data
    hipMemcpy(d_inputImage, inputImage, data_size, hipMemcpyHostToDevice);
    hipMemcpy(d_filter, filter, filterWidth * filterWidth * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(20, 30);
    dim3 numBlock(imageHeight / 20, imageWidth / 30);

    convolution<<<numBlock, blockSize>>>(d_inputImage, d_filter, img_result, filterWidth, imageHeight, imageWidth);

    // 等待 GPU 所有 thread 完成
    hipDeviceSynchronize();

    // 將 Device 的資料傳回給 Host
    hipMemcpy(outputImage, img_result, data_size, hipMemcpyDeviceToHost);

    // free memory
    hipFree(d_inputImage);
    hipFree(d_filter);
    hipFree(img_result);
}