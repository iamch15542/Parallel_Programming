#include "hip/hip_runtime.h"
#include "hostFE.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void convolution(float *inputImage, float *filter, float *outputImage, int filterWidth, int imageHeight, int imageWidth) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // col
    int j = blockIdx.y * blockDim.y + threadIdx.y; // row

    int sum = 0;
    int halfwidth = filterWidth / 2;
    // Apply the filter to the neighborhood
    for(int fi = -halfwidth; fi <= halfwidth; ++fi) {
        for(int fj = -halfwidth; fj <= halfwidth; ++fj) {
            if(i + fi >= 0 && i + fi < imageHeight && j + fj >= 0 && j + fj < imageWidth) {
                sum += inputImage[(i + fi) * imageWidth + j + fj] * filter[(fi + halfwidth) * filterWidth + fj + halfwidth];
            }
        }
    }
    outputImage[i * imageWidth + j] = sum;
}

void hostFE(int filterWidth, float *filter, int imageHeight, int imageWidth,
            float *inputImage, float *outputImage)
{
    // Declare the cuda memory
    int data_size = imageHeight * imageWidth * sizeof(float);

    float *d_inputImage;
    size_t pitch;
    hipMallocPitch((void **)&d_inputImage, &pitch, imageWidth * sizeof(float), imageHeight);

    float *d_filter;
    size_t pitch2;
    hipMallocPitch((void **)&d_filter, &pitch2, filterWidth * sizeof(float), filterWidth);

    float *img_result;
    size_t pitch3;
    hipMallocPitch((void **)&img_result, &pitch3, imageWidth * sizeof(float), imageHeight);
    printf("good\n")
    // copy data
    // hipMemcpy(d_inputImage, inputImage, data_size, hipMemcpyHostToDevice);
    // hipMemcpy(d_filter, filter, filterWidth * filterWidth * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy2D(d_inputImage, imageWidth * sizeof(float), inputImage, pitch, imageWidth * sizeof(float), imageHeight, hipMemcpyHostToDevice);
    hipMemcpy2D(d_filter, filterWidth * sizeof(float), filter, pitch2, filterWidth * sizeof(float), filterWidth, hipMemcpyHostToDevice);

    dim3 blockSize(20, 20);
    dim3 numBlock(imageHeight / 20, imageWidth / 20);

    convolution<<<numBlock, blockSize>>>(d_inputImage, d_filter, img_result, filterWidth, imageHeight, imageWidth);

    // 等待 GPU 所有 thread 完成
    hipDeviceSynchronize();

    // 將 Device 的資料傳回給 Host
    hipMemcpy(outputImage, img_result, data_size, hipMemcpyDeviceToHost);
    hipMemcpy2D(outputImage, imageWidth * sizeof(float), img_result, pitch3, imageWidth * sizeof(float), imageHeight, hipMemcpyDeviceToHost);

    // free memory
    hipFree(d_inputImage);
    hipFree(d_filter);
    hipFree(img_result);
}